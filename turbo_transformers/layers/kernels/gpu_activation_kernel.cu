#include "hip/hip_runtime.h"


#include <numeric>

#include "ide_macro.h"
#include "turbo_transformers/layers/kernels/gpu_activation_kernel.h"

namespace turbo_transformers {
namespace layers {
namespace kernels {
namespace {

template <typename T, ActivationType ActType>
__inline__ __device__ T ActivationOp(const T& x);

template <>
__inline__ __device__ float ActivationOp<float, ActivationType::Gelu>(
    const float& x) {
  float cdf =
      0.5f *
      (1.0f + tanhf((0.7978845608028654f * (x + 0.044715f * x * x * x))));
  return x * cdf;
}

template <>
__inline__ __device__ float ActivationOp<float, ActivationType::Tanh>(
    const float& x) {
  return tanhf(x);
}

template <>
__inline__ __device__ float ActivationOp<float, ActivationType::Relu>(
    const float& x) {
  return (x > 0) ? x : 0;
}

}  // namespace

template <typename T, ActivationType ActType>
static __global__ void add_bias_act(const T* bias, int batch_size,
                                    int feature_dim, T* out) {
  T val, reg_bias;

  int row_id;
  int elem_per_thread = (feature_dim + blockDim.x - 1) / blockDim.x;
  int tid = threadIdx.x;

  for (int i = 0; i < elem_per_thread; ++i) {
    int offset = i * blockDim.x + tid;
    if (offset < feature_dim) {
      reg_bias = bias[offset];
      row_id = blockIdx.x;
      val = out[offset + row_id * feature_dim] + reg_bias;
      out[offset + row_id * feature_dim] = ActivationOp<T, ActType>(val);
    }
  }
}

template <typename T, ActivationType ActType>
void GPUAddBiasActKernel(const T* bias_data, int64_t batch_size,
                         int64_t feature_dim, hipStream_t stream,
                         T* out_data) {
  dim3 grid(batch_size);
  int block_size = min(1024, (int)(feature_dim / 4));
  dim3 block(block_size);
  add_bias_act<T, ActType><<<grid, block, 0, stream>>>(bias_data, batch_size,
                                                       feature_dim, out_data);
}

template void GPUAddBiasActKernel<float, ActivationType::Gelu>(
    const float* bias_data, int64_t batch_size, int64_t feature_dim,
    hipStream_t stream, float* out_data);

template void GPUAddBiasActKernel<float, ActivationType::Tanh>(
    const float* bias_data, int64_t batch_size, int64_t feature_dim,
    hipStream_t stream, float* out_data);

template void GPUAddBiasActKernel<float, ActivationType::Relu>(
    const float* bias_data, int64_t batch_size, int64_t feature_dim,
    hipStream_t stream, float* out_data);

}  // namespace kernels
}  // namespace layers
}  // namespace turbo_transformers
